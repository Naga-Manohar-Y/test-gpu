#include "hip/hip_runtime.h"
#include "graph.h"
#include <hip/hip_runtime.h>
#include <>
#include <limits>

#define BLOCK_SIZE 32

__global__ void floyd_warshall_kernel(float* dist, int k, int n) {
    int i = blockIdx.y * blockDim.y + threadIdx.y;
    int j = blockIdx.x * blockDim.x + threadIdx.x;

    if (i < n && j < n) {
        int ij = i * n + j;
        int ik = i * n + k;
        int kj = k * n + j;
        float alt = dist[ik] + dist[kj];
        if (alt < dist[ij]) {
            dist[ij] = alt;
        }
    }
}

void Graph::computeAPSP() {
    hipError_t err;

    // Allocate host memory for APSP
    apsp = new float[n * n];
    
    // Initialize APSP matrix
    for (ui i = 0; i < n; i++) {
        for (ui j = 0; j < n; j++) {
            apsp[i * n + j] = (i == j) ? 0.0f : std::numeric_limits<float>::infinity();
        }
    }

    // Set initial distances based on graph structure and weights
    for (ui i = 0; i < n; i++) {
        ept start = neighbors_offset[i];
        ept end = neighbors_offset[i + 1];
        for (ept j = start; j < end; j++) {
            ui neighbor = neighbors[j];
            apsp[i * n + neighbor] = weights[j];
        }
    }

    // Allocate device memory for APSP if not already allocated
    if (d_apsp == nullptr) {
        err = hipMalloc(&d_apsp, n * n * sizeof(float));
        if (err != hipSuccess) {
            fprintf(stderr, "CUDA error (malloc): %s\n", hipGetErrorString(err));
            return;
        }
    }

    err = hipMemcpy(d_apsp, apsp, n * n * sizeof(float), hipMemcpyHostToDevice);
    if (err != hipSuccess) {
        fprintf(stderr, "CUDA error (memcpy to device): %s\n", hipGetErrorString(err));
        return;
    }

    // Set up grid and block dimensions
    dim3 block_dim(BLOCK_SIZE, BLOCK_SIZE);
    dim3 grid_dim((n + block_dim.x - 1) / block_dim.x, 
                  (n + block_dim.y - 1) / block_dim.y);

    // Run Floyd-Warshall algorithm
    for (ui k = 0; k < n; k++) {
        floyd_warshall_kernel<<<grid_dim, block_dim>>>(d_apsp, k, n);
        err = hipDeviceSynchronize();
        if (err != hipSuccess) {
            fprintf(stderr, "CUDA error (kernel): %s\n", hipGetErrorString(err));
            return;
        }
    }

    // Copy result back to host
    err = hipMemcpy(apsp, d_apsp, n * n * sizeof(float), hipMemcpyDeviceToHost);
    if (err != hipSuccess) {
        fprintf(stderr, "CUDA error (memcpy to host): %s\n", hipGetErrorString(err));
        return;
    }

    // Check for CUDA errors
    err = hipGetLastError();
    if (err != hipSuccess) {
        fprintf(stderr, "CUDA error: %s\n", hipGetErrorString(err));
    }
}