#include "hip/hip_runtime.h"
#include "graph.h"
#include <hip/hip_runtime.h>
#include <>
#include <limits>

__global__ void floyd_warshall_kernel(float* dist, int k, int n) {
    int i = blockIdx.y * blockDim.y + threadIdx.y;
    int j = blockIdx.x * blockDim.x + threadIdx.x;

    if (i < n && j < n) {
        int ij = i * n + j;
        int ik = i * n + k;
        int kj = k * n + j;
        float alt = dist[ik] + dist[kj];
        if (alt < dist[ij]) {
            dist[ij] = alt;
        }
    }
}

void Graph::computeAPSP() {
    // Allocate host memory for APSP
    apsp = new float[n * n];
    
    // Initialize APSP matrix
    for (ui i = 0; i < n; i++) {
        for (ui j = 0; j < n; j++) {
            apsp[i * n + j] = (i == j) ? 0.0f : std::numeric_limits<float>::infinity();
        }
    }

    // Set initial distances based on graph structure and weights
    for (ui i = 0; i < n; i++) {
        ept start = neighbors_offset[i];
        ept end = neighbors_offset[i + 1];
        for (ept j = start; j < end; j++) {
            ui neighbor = neighbors[j];
            apsp[i * n + neighbor] = weights[j];
        }
    }

    // Allocate device memory for APSP
    hipMalloc(&d_apsp, n * n * sizeof(float));
    hipMemcpy(d_apsp, apsp, n * n * sizeof(float), hipMemcpyHostToDevice);

    // Set up grid and block dimensions
    dim3 block_dim(32, 32);
    dim3 grid_dim((n + block_dim.x - 1) / block_dim.x, 
                  (n + block_dim.y - 1) / block_dim.y);

    // Run Floyd-Warshall algorithm
    for (ui k = 0; k < n; k++) {
        floyd_warshall_kernel<<<grid_dim, block_dim>>>(d_apsp, k, n);
        hipDeviceSynchronize();
    }

    // Copy result back to host
    hipMemcpy(apsp, d_apsp, n * n * sizeof(float), hipMemcpyDeviceToHost);

   

    // Check for CUDA errors
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        fprintf(stderr, "CUDA error: %s\n", hipGetErrorString(err));
    }
}